#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector_functions.hpp>

#include "color.h"
#include "float3.h"
#include "color.h"
#include "ray.h"
#include"hittable_list.h"
#include "used.h"
#include "camera.h"
#include "object.h"
#include "cudarand.h"
#include"scene.h"
#include "bvh.h"
#include "ray_color.h"

int main() {

    int dev = 0;
    hipSetDevice(dev);

    // World
    scene ourscene;

    // Camera
    auto cam = camera(LOOK_FROM, LOOK_AT, UP, FOV, ASPECT_RATIO).ToDevice();

    //Rand
    auto ourRand = cudaRand(image_width * image_height,0);

    // Render
    int nElem = image_width * image_height * 4;//RGBA
    int nByte = sizeof(float) * nElem;

    float* outHost = (float*)malloc(nByte);
    memset(outHost, 0, nByte);
    float* out;
    CHECK(hipMalloc((float**)&out, nByte));

    int nElemRay = image_width * image_height * 6;
    int nByteRay = sizeof(float) * nElemRay;

    float* rayHost = (float*)malloc(nByteRay);
    memset(rayHost, 0, nByteRay);
    float* rayDevice;
    CHECK(hipMalloc((float**)&rayDevice, nByteRay));

    ray_multisamples(out, ourscene, rayDevice, cam, ourRand.devStates);

    CHECK(hipMemcpy(outHost, out, nByte, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(rayHost, rayDevice, nByte, hipMemcpyDeviceToHost));

    drawPPM(outHost);

    hipFree(out);
    hipFree(rayDevice);

    hipFree(cam);
    hipFree(ourRand.devStates);
    free(outHost);

    hipDeviceReset();

    return 0;
}



